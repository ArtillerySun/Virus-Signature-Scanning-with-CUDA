#include "hip/hip_runtime.h"
#include "kseq/kseq.h"
#include "common.h"

#include <iostream>

const int P = 97;

__device__ inline bool check(char e, char f) {
    return e == 'N' || f == 'N' || e == f;
}

__global__ void get_phred(
    const int len, 
    const char* d_samples_quals, 
    unsigned char* d_sample_phred_score) {

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = id; i < len; i += stride) {
        d_sample_phred_score[i] = (unsigned char)d_samples_quals[i] - 33;
    }
}

__global__ void get_hash(
    const int len, 
    const unsigned char* d_samples_phred_score, 
    const size_t* d_samples_offset, 
    unsigned char* d_samples_hash) {

    int sample_idx = blockIdx.x;
    if (sample_idx >= len) return;

    size_t start_offset = d_samples_offset[sample_idx];
    size_t end_offset = d_samples_offset[sample_idx + 1];
    size_t sample_len = end_offset - start_offset;

    extern __shared__ int sh_sum[];
    int thread_sum = 0;

    for (size_t i = threadIdx.x; i < sample_len; i += blockDim.x) {
        thread_sum += d_samples_phred_score[start_offset + i];
    }
    sh_sum[threadIdx.x] = thread_sum;
    __syncthreads();

    for (int s = (blockDim.x >> 1); s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sh_sum[threadIdx.x] += sh_sum[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        d_samples_hash[sample_idx] = sh_sum[0] % P;
    }
}

struct TmpResult {
    size_t sample, signature;
    float match_score;
    int hash;
};

__global__ void matcher(
    const int SAMPLES_SIZE,
    const int SIGNATURES_SIZE,
    const unsigned char* d_samples_hash, 
    const unsigned char* d_samples_phred_score, 
    const size_t* d_samples_offset, 
    const char* d_samples_seqs, 
    const size_t* d_signatures_offset, 
    const char* d_signatures_seqs, 
    TmpResult* tmpResult) {
    
    size_t sample_idx = blockIdx.x ;
    size_t signature_idx = blockIdx.y;

    if (sample_idx >= SAMPLES_SIZE) return;
    if (signature_idx >= SIGNATURES_SIZE) return;
    
    size_t sample_start_offset = d_samples_offset[sample_idx];
    size_t sample_end_offset = d_samples_offset[sample_idx + 1];
    size_t sample_len = sample_end_offset - sample_start_offset;

    size_t signature_start_offset = d_signatures_offset[signature_idx];
    size_t signature_end_offset = d_signatures_offset[signature_idx + 1];
    size_t signature_len = signature_end_offset - signature_start_offset;

    if (signature_len > sample_len) return;

    size_t search_space = sample_len - signature_len + 1;


    extern __shared__ char sh_mem[];
    float* sh_max = (float*)(sh_mem);
    char* sh_signature_seq = (char*)&sh_max[blockDim.x];

    for (size_t i = threadIdx.x; i < signature_len; i += blockDim.x) {
        sh_signature_seq[i] = d_signatures_seqs[signature_start_offset + i];
    }

    sh_max[threadIdx.x] = -1.0;
    __syncthreads();

    for (size_t i = threadIdx.x; i < search_space; i += blockDim.x) {
        float sum = 0;
        bool flag = true;
        for (size_t j = 0; j < signature_len; j++) {
            if (!check(d_samples_seqs[sample_start_offset + i + j], sh_signature_seq[j])) {
                flag = false;
                break;
            }
            sum += d_samples_phred_score[sample_start_offset + i + j];
        }
        if (flag) {
            sh_max[threadIdx.x] = fmaxf(sh_max[threadIdx.x], sum/signature_len);
        }
    }
    __syncthreads();

    for (int s = (blockDim.x >> 1); s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sh_max[threadIdx.x] = fmaxf(sh_max[threadIdx.x], sh_max[threadIdx.x + s]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        if (sh_max[0] >= 0) {
            tmpResult[sample_idx * SIGNATURES_SIZE + signature_idx] = {sample_idx, signature_idx, sh_max[0], d_samples_hash[sample_idx]};
        }
    }
    
}

void runMatcher(const std::vector<klibpp::KSeq>& samples, const std::vector<klibpp::KSeq>& signatures, std::vector<MatchResult>& matches) {
    const int SIGNATURES_SIZE = signatures.size();

    hipStream_t stream;
    hipStreamCreate(&stream);

    size_t total_signatures_len = 0;
    for (const auto& signature : signatures) {
        total_signatures_len += signature.seq.length();
    }

    size_t* h_signatures_offset_pinned;
    char* h_signatures_seqs_pinned;
    hipHostAlloc(&h_signatures_seqs_pinned, total_signatures_len * sizeof(char), hipHostMallocDefault);
    hipHostAlloc(&h_signatures_offset_pinned, (SIGNATURES_SIZE + 1) * sizeof(size_t), hipHostMallocDefault);

    size_t max_signature_len = 0;
    size_t current_offset = 0;

    h_signatures_offset_pinned[0] = 0;
    for (int i = 0; i < SIGNATURES_SIZE; i++) {
        const auto& signature = signatures[i];
        size_t len = signature.seq.length();
        max_signature_len = std::max(max_signature_len, len);
        memcpy(h_signatures_seqs_pinned + current_offset, signature.seq.c_str(), len);
        
        current_offset += len;
        h_signatures_offset_pinned[i + 1] = current_offset;
    }

    // signatures on device
    char *d_signatures_seqs;
    size_t *d_signatures_offset;

    // allocate memory for signatures on device
    hipMalloc(&d_signatures_seqs, total_signatures_len * sizeof(char));
    hipMalloc(&d_signatures_offset, (SIGNATURES_SIZE + 1) * sizeof(size_t));

    // copy signatures to device
    hipMemcpyAsync(d_signatures_seqs, h_signatures_seqs_pinned, total_signatures_len * sizeof(char), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_signatures_offset, h_signatures_offset_pinned, (SIGNATURES_SIZE + 1) * sizeof(size_t), hipMemcpyHostToDevice, stream);
    
    const int BATCH_SIZE = 1024; // should be 2-power

    for (int batch_start = 0; batch_start < samples.size(); batch_start += BATCH_SIZE) {

        int batch_end = std::min(batch_start + BATCH_SIZE, (int)samples.size());
        const int SAMPLES_SIZE = batch_end - batch_start;

        auto samples_begin = samples.begin() + batch_start;
        auto samples_end = samples.begin() + batch_end;

        size_t total_samples_len = 0;
        for (auto sample = samples_begin; sample != samples_end; sample++) {
            total_samples_len += sample->seq.length();
        }

        char *h_samples_seqs_pinned, *h_samples_quals_pinned;
        size_t *h_samples_offset_pinned;
        TmpResult* h_tmpResult_pinned;

        hipHostAlloc(&h_samples_seqs_pinned, total_samples_len * sizeof(char), hipHostMallocDefault);
        hipHostAlloc(&h_samples_quals_pinned, total_samples_len * sizeof(char), hipHostMallocDefault);
        hipHostAlloc(&h_samples_offset_pinned, (SAMPLES_SIZE + 1) * sizeof(size_t), hipHostMallocDefault);
        hipHostAlloc(&h_tmpResult_pinned, SAMPLES_SIZE * SIGNATURES_SIZE * sizeof(TmpResult), hipHostMallocDefault);

        size_t current_offset = 0;

        size_t cnt = 0;
        h_samples_offset_pinned[cnt] = 0;
        for (auto sample = samples_begin; sample != samples_end; sample++) {
            size_t len = sample->seq.length();
            memcpy(h_samples_seqs_pinned + current_offset, sample->seq.c_str(), len);
            memcpy(h_samples_quals_pinned + current_offset, sample->qual.c_str(), len);
            
            current_offset += len;
            h_samples_offset_pinned[++cnt] = current_offset;
        }

        for (int i = 0; i < SAMPLES_SIZE * SIGNATURES_SIZE; i++) {
            h_tmpResult_pinned[i].match_score = -1.0f;
        }

        // samples on device
        char *d_samples_quals, *d_samples_seqs;
        unsigned char* d_samples_phred_score, *d_samples_hash;
        size_t *d_samples_offset;

        TmpResult* d_tmpResult;

        // allocate memory for samples on device
        hipMalloc(&d_samples_seqs, total_samples_len * sizeof(char));
        hipMalloc(&d_samples_offset, (SAMPLES_SIZE + 1) * sizeof(size_t));
        hipMalloc(&d_samples_hash, SAMPLES_SIZE * sizeof(unsigned char));
        hipMalloc(&d_samples_quals, total_samples_len * sizeof(char));
        hipMalloc(&d_samples_phred_score, total_samples_len * sizeof(unsigned char));
        hipMalloc(&d_tmpResult, SAMPLES_SIZE * SIGNATURES_SIZE * sizeof(TmpResult));

        // copy samples to device
        hipMemcpyAsync(d_samples_seqs, h_samples_seqs_pinned, total_samples_len * sizeof(char), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_samples_offset, h_samples_offset_pinned, (SAMPLES_SIZE + 1) * sizeof(size_t), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_samples_quals, h_samples_quals_pinned, total_samples_len * sizeof(char), hipMemcpyHostToDevice, stream);

        hipMemcpyAsync(d_tmpResult, h_tmpResult_pinned, SAMPLES_SIZE * SIGNATURES_SIZE * sizeof(TmpResult), hipMemcpyHostToDevice, stream);

        // calculate phred score with the +33 version
        int blk_size = 256;
        int blk_num = (total_samples_len + blk_size - 1) / blk_size;
        get_phred<<<blk_num, blk_size, 0, stream>>>(total_samples_len, d_samples_quals, d_samples_phred_score);
        
        // calculate hash value for all samples
        blk_num = SAMPLES_SIZE;
        get_hash<<<blk_num, blk_size, blk_size * sizeof(int), stream>>>(SAMPLES_SIZE, d_samples_phred_score, d_samples_offset, d_samples_hash);

        // match the signatures
        dim3 grid(SAMPLES_SIZE, SIGNATURES_SIZE);
        matcher<<<grid, blk_size, max_signature_len * sizeof(char) + blk_size * sizeof(float), stream>>>(SAMPLES_SIZE, SIGNATURES_SIZE, d_samples_hash, d_samples_phred_score, d_samples_offset, d_samples_seqs, d_signatures_offset, d_signatures_seqs, d_tmpResult);
        
        hipMemcpyAsync(h_tmpResult_pinned, d_tmpResult, SAMPLES_SIZE * SIGNATURES_SIZE * sizeof(TmpResult), hipMemcpyDeviceToHost);
        hipStreamSynchronize(stream);

        for (int i = 0; i < SAMPLES_SIZE; i ++) {
            for (int j = 0; j < SIGNATURES_SIZE; j ++) {
                auto& cur = h_tmpResult_pinned[i * SIGNATURES_SIZE + j];
                if (cur.match_score >= 0) {
                    matches.push_back({samples[batch_start + i].name, signatures[j].name, cur.match_score, cur.hash});
                }
            }
        }
        
        // release all the allocated memory on device
        hipFree(d_samples_seqs);
        hipFree(d_samples_offset);
        hipFree(d_samples_hash);
        hipFree(d_samples_quals);
        hipFree(d_samples_phred_score);
        hipFree(d_tmpResult);

        hipHostFree(h_samples_seqs_pinned);
        hipHostFree(h_samples_quals_pinned);
        hipHostFree(h_samples_offset_pinned);
        hipHostFree(h_tmpResult_pinned);
    }

    hipStreamDestroy(stream);

    hipFree(d_signatures_seqs);
    hipFree(d_signatures_offset);

    hipHostFree(h_signatures_seqs_pinned);
    hipHostFree(h_signatures_offset_pinned);
}
